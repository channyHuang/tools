#include "hip/hip_runtime.h"
# include "AddVectors.h"
//# include <mex.h>

__global__ void addVectorsMask(float* A, float* B, float* C,int size)
{
        int i=blockIdx.x;
        if(i>=size)
        return;

        C[i]=A[i]+B[i];
}


void addVectors(float* A, float* B, float* C,int size)
{
        float *devPtrA=0;
        float *devPtrB=0;
        float *devPtrC=0;


        hipMalloc(&devPtrA,sizeof(float)*size);
        hipMalloc(&devPtrB,sizeof(float)*size);
        hipMalloc(&devPtrC,sizeof(float)*size);

        hipMemcpy(devPtrA,A,sizeof(float)*size,hipMemcpyHostToDevice);
        hipMemcpy(devPtrB,B,sizeof(float)*size,hipMemcpyHostToDevice);


        addVectorsMask<<<size,1>>>(devPtrA,devPtrB,devPtrC,size);

        hipMemcpy(C,devPtrC,sizeof(float)*size,hipMemcpyDeviceToHost);

        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);

}

